#include "hip/hip_runtime.h"
/* Udacity Homework 3
HDR Tone-mapping

Background HDR
==============

A High Dynamic Range (HDR) image contains a wider variation of intensity
and color than is allowed by the RGB format with 1 byte per channel that we
have used in the previous assignment.

To store this extra information we use single precision floating point for
each channel.  This allows for an extremely wide range of intensity values.

In the image for this assignment, the inside of church with light coming in
through stained glass windows, the raw input floating point values for the
channels range from 0 to 275.  But the mean is .41 and 98% of the values are
less than 3!  This means that certain areas (the windows) are extremely bright
compared to everywhere else.  If we linearly map this [0-275] range into the
[0-255] range that we have been using then most values will be mapped to zero!
The only thing we will be able to see are the very brightest areas - the
windows - everything else will appear pitch black.

The problem is that although we have cameras capable of recording the wide
range of intensity that exists in the real world our monitors are not capable
of displaying them.  Our eyes are also quite capable of observing a much wider
range of intensities than our image formats / monitors are capable of
displaying.

Tone-mapping is a process that transforms the intensities in the image so that
the brightest values aren't nearly so far away from the mean.  That way when
we transform the values into [0-255] we can actually see the entire image.
There are many ways to perform this process and it is as much an art as a
science - there is no single "right" answer.  In this homework we will
implement one possible technique.

Background Chrominance-Luminance
================================

The RGB space that we have been using to represent images can be thought of as
one possible set of axes spanning a three dimensional space of color.  We
sometimes choose other axes to represent this space because they make certain
operations more convenient.

Another possible way of representing a color image is to separate the color
information (chromaticity) from the brightness information.  There are
multiple different methods for doing this - a common one during the analog
television days was known as Chrominance-Luminance or YUV.

We choose to represent the image in this way so that we can remap only the
intensity channel and then recombine the new intensity values with the color
information to form the final image.

Old TV signals used to be transmitted in this way so that black & white
televisions could display the luminance channel while color televisions would
display all three of the channels.


Tone-mapping
============

In this assignment we are going to transform the luminance channel (actually
the log of the luminance, but this is unimportant for the parts of the
algorithm that you will be implementing) by compressing its range to [0, 1].
To do this we need the cumulative distribution of the luminance values.

Example
-------

input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
min / max / range: 0 / 9 / 9

histo with 3 bins: [4 7 3]

cdf : [4 11 14]


Your task is to calculate this cumulative distribution by following these
steps.

*/

#include "utils.h"


// GPU Kernels *************

__global__ void find_max_kernel(float * d_out, const float * d_in)
{
	// sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
	extern __shared__ float sdata[];

	const int threadGId = blockIdx.x * blockDim.x + threadIdx.x;
	const int threadLId = threadIdx.x;

	// load shared mem from global mem
	sdata[threadLId] = d_in[threadGId];
	__syncthreads();            // make sure entire block is loaded!

								// do reduction in shared mem
	for (unsigned int blockHalfSize = blockDim.x / 2; blockHalfSize > 0; blockHalfSize >>= 1) {
		if (threadLId < blockHalfSize) {
			sdata[threadLId] = max(sdata[threadLId], sdata[threadLId + blockHalfSize]);
		}
		__syncthreads();        // make sure all adds at one stage are done!
	}

	// only thread 0 writes result for this block back to global mem
	if (threadLId == 0)
	{
		d_out[blockIdx.x] = sdata[0];
	}
}

__global__ void find_min_kernel(float * d_out, const float * d_in)
{
	// sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
	extern __shared__ float sdata[];

	const int threadGId = blockIdx.x * blockDim.x + threadIdx.x;
	const int threadLId = threadIdx.x;

	// load shared mem from global mem
	sdata[threadLId] = d_in[threadGId];
	__syncthreads();            // make sure entire block is loaded!

								// do reduction in shared mem
	for (unsigned int blockHalfSize = blockDim.x / 2; blockHalfSize > 0; blockHalfSize >>= 1) {
		if (threadLId < blockHalfSize) {
			sdata[threadLId] = min(sdata[threadLId], sdata[threadLId + blockHalfSize]);
		}
		__syncthreads();        // make sure all adds at one stage are done!
	}

	// only thread 0 writes result for this block back to global mem
	if (threadLId == 0)
	{
		d_out[blockIdx.x] = sdata[0];
	}
}

__global__ void calc_histogram_kernel(const float* const d_in,
	float* d_minArray,
	float* d_maxArray,
	size_t* d_numBins,
	unsigned int * d_bincount) {

	const int threadGId = blockIdx.x * blockDim.x + threadIdx.x;

	const float range = *d_maxArray - *d_minArray;
	float item = d_in[threadGId];

	int bin = (item - *d_minArray) / range * *d_numBins;
	//printf("BlockDim: %d BlockInd: %d ThreadG: %d Item: %f Bin: %d\n", blockDim.x, blockIdx.x, threadGId, item, bin);

	atomicAdd(&(d_bincount[bin]), 1);
}

__global__ void calc_cdf_kernel(unsigned int* const d_cdf,
	size_t ArraySize) {

	const int threadGId = blockIdx.x * blockDim.x + threadIdx.x;

	for (int s = 1; s <= ArraySize; s *= 2) {
		int spot = threadGId - s;

		unsigned int val = 0;
		if (spot >= 0)
			val = d_cdf[spot];
		__syncthreads();
		if (spot >= 0)
			d_cdf[threadGId] += val;
		__syncthreads();
	}
}




void find_max(const float* const d_logLuminance,
	float &max_logLum,
	const size_t ArraySize) {

	// declare GPU memory pointers
	float *d_intermediate, *d_out;

	size_t ARRAY_BYTES = sizeof(float)*ArraySize;

	// Allocate memory on GPU
	checkCudaErrors(hipMalloc((void **)&d_intermediate, ARRAY_BYTES / 2));
	checkCudaErrors(hipMalloc((void **)&d_out, sizeof(float)));


	const int maxThreadsPerBlock = 1024;
	int threads = maxThreadsPerBlock;
	int blocks = ArraySize / maxThreadsPerBlock;

	// Launch the first kernel to find the min and max
	find_max_kernel << <blocks, threads, threads * sizeof(float) >> > (d_intermediate, d_logLuminance);

	// Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
	// launching your kernel to make sure that you didn't make any mistakes.
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	threads = blocks;
	blocks = 1;
	// Launch the second kernel to find the min and max
	find_max_kernel << <blocks, threads, threads * sizeof(float) >> > (d_out, d_intermediate);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	// Get max value from GPU memory
	checkCudaErrors(hipMemcpy(&max_logLum, d_out, sizeof(float), hipMemcpyDeviceToHost));

	// free GPU memory allocation
	checkCudaErrors(hipFree(d_intermediate));
	checkCudaErrors(hipFree(d_out));

}

void find_min(const float* const d_logLuminance,
	float &min_logLum,
	const size_t ArraySize) {

	// declare GPU memory pointers
	float *d_intermediate, *d_out;

	size_t ARRAY_BYTES = sizeof(float)*ArraySize;

	// Allocate memory on GPU
	checkCudaErrors(hipMalloc((void **)&d_intermediate, ARRAY_BYTES / 2));
	checkCudaErrors(hipMalloc((void **)&d_out, sizeof(float)));


	const int maxThreadsPerBlock = 1024;
	int threads = maxThreadsPerBlock;
	int blocks = ArraySize / maxThreadsPerBlock;

	// Launch the first kernel to find the min and max
	find_min_kernel << <blocks, threads, threads * sizeof(float) >> > (d_intermediate, d_logLuminance);

	// Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
	// launching your kernel to make sure that you didn't make any mistakes.
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	threads = blocks;
	blocks = 1;
	// Launch the second kernel to find the min and max
	find_min_kernel << <blocks, threads, threads * sizeof(float) >> > (d_out, d_intermediate);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	// Get max value from GPU memory
	checkCudaErrors(hipMemcpy(&min_logLum, d_out, sizeof(float), hipMemcpyDeviceToHost));

	// free GPU memory allocation
	checkCudaErrors(hipFree(d_intermediate));
	checkCudaErrors(hipFree(d_out));

}

void calc_histogram(const float* const d_logLuminance,
	unsigned int *h_bincount,
	float &min_logLum,
	float &max_logLum,
	const size_t numBins,
	const size_t ArraySize) {

	// declare GPU memory pointers
	float *d_minArray, *d_maxArray;
	size_t *d_numBins;
	unsigned int *d_bincount;

	// Allocate memory on GPU
	checkCudaErrors(hipMalloc((void **)&d_minArray, sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&d_maxArray, sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&d_numBins, sizeof(size_t)));
	checkCudaErrors(hipMalloc((void **)&d_bincount, sizeof(unsigned int)*numBins));

	// Copy from host to device
	checkCudaErrors(hipMemcpy(d_minArray, &min_logLum, sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_maxArray, &max_logLum, sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_numBins, &numBins, sizeof(size_t), hipMemcpyHostToDevice));

	const int maxThreadsPerBlock = 1024;
	int threads = maxThreadsPerBlock;
	int blocks = ArraySize / maxThreadsPerBlock;

	calc_histogram_kernel << <blocks, threads >> > (d_logLuminance, d_minArray, d_maxArray, d_numBins, d_bincount);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());


	// Get bincount value from GPU memory
	checkCudaErrors(hipMemcpy(h_bincount, d_bincount, sizeof(unsigned int)*numBins, hipMemcpyDeviceToHost));

	// free GPU memory allocation
	checkCudaErrors(hipFree(d_minArray));
	checkCudaErrors(hipFree(d_maxArray));
	checkCudaErrors(hipFree(d_numBins));
	checkCudaErrors(hipFree(d_bincount));

}


void calc_cdf(unsigned int *h_bincount,
	const size_t numBins,
	size_t ArraySize,
	unsigned int* const d_cdf) {

	unsigned int * d_bincount;

	// Allocate memory on GPU
	checkCudaErrors(hipMalloc((void **)&d_bincount, sizeof(unsigned int)*numBins));

	// Copy from host to device
	checkCudaErrors(hipMemcpy(d_cdf, h_bincount, sizeof(unsigned int)*numBins, hipMemcpyHostToDevice));

	const int maxThreadsPerBlock = 1024;
	int threads = maxThreadsPerBlock;
	int blocks = ArraySize / maxThreadsPerBlock;

	calc_cdf_kernel << <blocks, threads >> > (d_cdf, ArraySize);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}




void your_histogram_and_prefixsum(const float* const d_logLuminance,
	unsigned int* const d_cdf,
	float &min_logLum,
	float &max_logLum,
	const size_t numRows,
	const size_t numCols,
	const size_t numBins)
{
	//TODO
	/*Here are the steps you need to implement
	1) find the minimum and maximum value in the input logLuminance channel
	store in min_logLum and max_logLum
	2) subtract them to find the range
	3) generate a histogram of all the values in the logLuminance channel using
	the formula: bin = (lum[i] - lumMin) / lumRange * numBins
	4) Perform an exclusive scan (prefix sum) on the histogram to get
	the cumulative distribution of luminance values (this should go in the
	incoming d_cdf pointer which already has been allocated for you)       */


	const size_t arraySize = numRows*numCols;

	find_max(d_logLuminance, max_logLum, arraySize);
	find_min(d_logLuminance, min_logLum, arraySize);

	float range = max_logLum - min_logLum;

	printf("Max: %f Min: %f Range: %f\n", max_logLum, min_logLum, range);

	// Allocate memory for histogram
	unsigned int *h_bincount = (unsigned int *)malloc(sizeof(unsigned int)*numBins);

	calc_histogram(d_logLuminance, h_bincount, min_logLum, max_logLum, numBins, arraySize);

	calc_cdf(h_bincount, numBins, arraySize, d_cdf);


	/*unsigned int *h_cdf = (unsigned int *)malloc(sizeof(unsigned int)*numBins);
	hipMemcpy(h_cdf, d_cdf, sizeof(unsigned int) * 100, hipMemcpyDeviceToHost);

	for (int i = 0; i < 100; i++)
		printf("Hst out %d\n", h_bincount[i]);

	for (int i = 0; i < 100; i++)
		printf("Cdf out %d\n", h_cdf[i]);*/

}
