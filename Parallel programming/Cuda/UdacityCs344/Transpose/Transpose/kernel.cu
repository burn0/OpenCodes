#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "gputimer.h"

#include <stdio.h>

const int N = 512;		// matrix size is NxN
const int K = 16;	    	// TODO, set K to the correct value and tile size will be KxK


							// to be launched with one thread per element, in KxK threadblocks
							// thread (x,y) in grid writes element (i,j) of output matrix 
__global__ void
transpose_parallel_per_element(float in[], float out[])
{
	//TODO
	const int thread_2D_posX = blockIdx.x * blockDim.x + threadIdx.x;
	const int thread_2D_posY  = blockIdx.y * blockDim.y + threadIdx.y;

	out[thread_2D_posY + thread_2D_posX*N] = in[thread_2D_posX + thread_2D_posY*N]; // out(j,i) = in(i,j)

}


// to be launched on a single thread
__global__ void
transpose_serial(float in[], float out[])
{
	for (int j = 0; j < N; j++)
		for (int i = 0; i < N; i++)
			out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched with one thread per row of output matrix
__global__ void
transpose_parallel_per_row(float in[], float out[])
{
	int i = threadIdx.x;

	for (int j = 0; j < N; j++)
		out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}






//The following functions and kernels are for your reference
void transpose_CPU(float in[], float out[])
{
	for (int j = 0; j < N; j++)
		for (int i = 0; i < N; i++)
			out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// fill a matrix with sequential numbers in the range 0..N-1
void fill_matrix(float *mat)
{
	for (int j = 0; j < N * N; j++)
		mat[j] = (float)j;
}

int compare_matrices(float *gpu, float *ref)
{
	int result = 0;

	for (int j = 0; j < N; j++)
		for (int i = 0; i < N; i++)
			if (ref[i + j*N] != gpu[i + j*N])
			{
				// printf("reference(%d,%d) = %f but test(%d,%d) = %f\n",
				// i,j,ref[i+j*N],i,j,test[i+j*N]);
				result = 1;
			}
	return result;
}






int main(int argc, char **argv)
{
	int numbytes = N * N * sizeof(float);

	float *in = (float *)malloc(numbytes);
	float *out = (float *)malloc(numbytes);
	float *gold = (float *)malloc(numbytes);

	fill_matrix(in);
	transpose_CPU(in, gold);

	float *d_in, *d_out;

	hipMalloc(&d_in, numbytes);
	hipMalloc(&d_out, numbytes);
	hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);

	GpuTimer timer;


	// Serial
	/*timer.Start();
	transpose_serial << <1, 1 >> >(d_in, d_out);
	timer.Stop();
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_serial: %g ms.\nVerifying transpose...%s\n",
		timer.Elapsed(), compare_matrices(out, gold) ? "Failed" : "Success");*/

	// 1 Thread per row
	timer.Start();
	transpose_parallel_per_row << <1, N >> >(d_in, d_out);
	timer.Stop();
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_row: %g ms.\nVerifying transpose...%s\n",
		timer.Elapsed(), compare_matrices(out, gold) ? "Failed" : "Success");


	/*
	* Now time each kernel and verify that it produces the correct result.
	*
	* To be really careful about benchmarking purposes, we should run every kernel once
	* to "warm" the system and avoid any compilation or code-caching effects, then run
	* every kernel 10 or 100 times and average the timings to smooth out any variance.
	* But this makes for messy code and our goal is teaching, not detailed benchmarking.
	*/

	dim3 threads(K, K);	// TODO, you need to define the correct threads per block
	dim3 blocks(N/K, N/K); // TODO, you need to define the correct blocks per grid

	timer.Start();
	transpose_parallel_per_element << <blocks, threads >> >(d_in, d_out);
	timer.Stop();
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element: %g ms.\nVerifying transpose...%s\n",
		timer.Elapsed(), compare_matrices(out, gold) ? "Failed" : "Success");

	hipFree(d_in);
	hipFree(d_out);
}